#include "hip/hip_runtime.h"


#ifndef __INTELLISENSE__
#define KERNEL_ARGS2(grid, block)                 <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem)         <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif

extern "C" {
	__global__ void matrixMulEx(int* a, int* b, int* c, int N)
	{
		//Calculate the global row and column for each thread
		int row = blockIdx.y * blockDim.y + threadIdx.y;
		int col = blockIdx.x * blockDim.x + threadIdx.x;

		//Bounds check for matrix
		if (row < N && col < N)
		{
			//Accumulate a partial result
			int temp = 0;
			for (int i = 0; i < N; i++)
			{
				temp += a[row * N + i] * b[i * N + col];
			}

			//Write back result
			c[row * N + col] = temp;
		}
	}
}
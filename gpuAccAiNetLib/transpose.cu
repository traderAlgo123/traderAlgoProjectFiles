#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "hip/hip_runtime.h"


extern "C" {
	__global__ void transpose(double* idata, double* odata, int width, int height)
	{
        unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
        unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

        if (xIndex < width && yIndex < height)
        {
            unsigned int index_in = xIndex + width * yIndex;
            unsigned int index_out = yIndex + height * xIndex;
            odata[index_out] = idata[index_in];
        }
	}
}
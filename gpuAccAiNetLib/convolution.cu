#include "hip/hip_runtime.h"


extern "C" {
	__global__ void convolution(double* pricesBlock, double* sizesBlock,
		 double* pricesKernel, double* sizesKernel, double* pricesBlock2,
		 double* sizesBlock2, double* pricesKernel2, double* sizesKernel2,
		 double* resVal, int N)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < N)
		{
			resVal[i] = (pricesBlock[i] * pricesKernel[i]) + (sizesBlock[i] * sizesKernel[i]) + (pricesBlock2[i] *
				pricesKernel2[i]) + (sizesBlock2[i] * sizesKernel2[i]);
		}
	}
}

extern "C" {
	__global__ void convolution2(double* featuresBlock, double* featuresKernel, double* featuresBlock2,
        double* featuresKernel2, double* resVal, int N)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < N)
		{
			resVal[i] = (featuresBlock[i] * featuresKernel[i]) + (featuresBlock2[i] * featuresKernel2[i]);
		}
	}
}

extern "C" {
	__global__ void convolutionBackProp(double* derBlock, double* inputBlock, double* resVal, int N)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;

		if (i < N)
		{
			resVal[i] = (derBlock[i] * inputBlock[i]);
		}
	}
}
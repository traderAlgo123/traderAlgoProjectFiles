#include "hip/hip_runtime.h"


#ifndef __INTELLISENSE__
#define KERNEL_ARGS2(grid, block)                 <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem)         <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif

extern "C" {
    __global__ void matrixMul(double* a, double* b, double* c, int m, int n, int k)
    {
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        double sum = 0;
        if (col < k && row < m)
        {
            for (int i = 0; i < n; i++)
            {
                sum += a[row * n + i] * b[i * k + col];
            }
            c[row * k + col] = sum;
        }
    }
}
